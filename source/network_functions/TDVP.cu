#include "hip/hip_runtime.h"
// ***********************************************************
// *       This is an automatically generated file.          *
// *       For editing, please use the source file:          *
// TDVP.cu.template
// ***********************************************************

#ifndef LEAN_AND_MEAN

#include "network_functions/TDVP.hpp"
#include "quantum_states.hpp"
#include "ensembles.hpp"


namespace ann_on_gpu {


template<typename Psi_t, typename Ensemble>
void TDVP::eval(const Operator& op, const Psi_t& psi, Ensemble& ensemble) {
    this->E_local_ar.clear();
    this->O_k_ar.clear();
    this->S_matrix.clear();
    this->F_vector.clear();
    this->prob_ratio.clear();

    auto num_params = this->F_vector.size();
    auto op_kernel = op.kernel();
    auto psi_kernel = psi.kernel();
    auto psi_ref_kernel = psi.psi_ref.kernel();
    auto E_local_ptr = this->E_local_ar.data();
    auto O_k_ptr = this->O_k_ar.data();
    auto S_ptr = this->S_matrix.data();
    auto F_ptr = this->F_vector.data();
    auto prob_ratio_ptr = this->prob_ratio.data();

    using PsiRef = typename Psi_t::PsiRef;

    ensemble.foreach(
        psi.psi_ref,
        [=] __device__ __host__ (
            const unsigned int index,
            const typename Ensemble::Basis_t& configuration,
            const typename PsiRef::dtype log_psi_ref,
            typename PsiRef::Payload& payload_ref,
            const typename PsiRef::real_dtype weight
        ) {
            #include "cuda_kernel_defines.h"

            SHARED complex_t                   log_psi;
            SHARED typename Psi_t::Payload     payload;

            SHARED complex_t local_energy;
            psi_kernel.log_psi_s(log_psi, configuration, payload);
            op_kernel.local_energy(local_energy, psi_kernel, configuration, log_psi, payload);

            SHARED double prob_ratio;

            SINGLE {
                prob_ratio = exp(2.0 * (log_psi.real() - log_psi_ref.real()));
                generic_atomicAdd(prob_ratio_ptr, weight * prob_ratio);
                generic_atomicAdd(E_local_ptr, weight * prob_ratio * local_energy);
            }

            psi_kernel.init_payload(payload, configuration);
            psi_kernel.foreach_O_k(
                configuration,
                payload,
                [&](const unsigned int k, const complex_t& O_k) {
                    generic_atomicAdd(&O_k_ptr[k], weight * prob_ratio * O_k);
                    generic_atomicAdd(&F_ptr[k], weight * prob_ratio * local_energy * conj(O_k));

                    for(auto k_prime = 0u; k_prime < psi_kernel.num_params; k_prime++) {
                        generic_atomicAdd(
                            &S_ptr[k * num_params + k_prime],
                            weight * prob_ratio * conj(O_k) * psi_kernel.get_O_k(k_prime, payload)
                        );
                    }
                }
            );
        }
    );

    this->E_local_ar.update_host();
    this->O_k_ar.update_host();
    this->S_matrix.update_host();
    this->F_vector.update_host();
    this->prob_ratio.update_host();

    this->E_local_ar.front() /= this->prob_ratio.front();
    for(auto k = 0u; k < num_params; k++) {
        this->O_k_ar[k] /= this->prob_ratio.front();
        this->F_vector[k] /= this->prob_ratio.front();

        for(auto k_prime = 0u; k_prime < num_params; k_prime++) {
            this->S_matrix[k * num_params + k_prime] /= this->prob_ratio.front();
        }
    }

    for(auto k = 0u; k < num_params; k++) {
        for(auto k_prime = 0u; k_prime < num_params; k_prime++) {
            this->S_matrix[k * num_params + k_prime] -= (
                conj(this->O_k_ar[k]) * this->O_k_ar[k_prime]
            );
        }

        this->F_vector[k] -= this->E_local_ar.front() * conj(this->O_k_ar[k]);
    }
}


#if defined(ENABLE_MONTE_CARLO) && defined(ENABLE_SPINS) && defined(ENABLE_PSI_CLASSICAL)
template void TDVP::eval(const Operator&, const PsiClassicalFP<1u>&, MonteCarlo_tt<Spins>&);
#endif
#if defined(ENABLE_MONTE_CARLO) && defined(ENABLE_SPINS) && defined(ENABLE_PSI_CLASSICAL)
template void TDVP::eval(const Operator&, const PsiClassicalFP<2u>&, MonteCarlo_tt<Spins>&);
#endif
#if defined(ENABLE_MONTE_CARLO) && defined(ENABLE_SPINS) && defined(ENABLE_PSI_CLASSICAL) && defined(ENABLE_PSI_CLASSICAL_ANN)
template void TDVP::eval(const Operator&, const PsiClassicalANN<1u>&, MonteCarlo_tt<Spins>&);
#endif
#if defined(ENABLE_MONTE_CARLO) && defined(ENABLE_SPINS) && defined(ENABLE_PSI_CLASSICAL) && defined(ENABLE_PSI_CLASSICAL_ANN)
template void TDVP::eval(const Operator&, const PsiClassicalANN<2u>&, MonteCarlo_tt<Spins>&);
#endif
#if defined(ENABLE_MONTE_CARLO) && defined(ENABLE_PAULIS) && defined(ENABLE_PSI_CLASSICAL)
template void TDVP::eval(const Operator&, const PsiClassicalFP<1u>&, MonteCarlo_tt<PauliString>&);
#endif
#if defined(ENABLE_MONTE_CARLO) && defined(ENABLE_PAULIS) && defined(ENABLE_PSI_CLASSICAL)
template void TDVP::eval(const Operator&, const PsiClassicalFP<2u>&, MonteCarlo_tt<PauliString>&);
#endif
#if defined(ENABLE_MONTE_CARLO) && defined(ENABLE_PAULIS) && defined(ENABLE_PSI_CLASSICAL) && defined(ENABLE_PSI_CLASSICAL_ANN)
template void TDVP::eval(const Operator&, const PsiClassicalANN<1u>&, MonteCarlo_tt<PauliString>&);
#endif
#if defined(ENABLE_MONTE_CARLO) && defined(ENABLE_PAULIS) && defined(ENABLE_PSI_CLASSICAL) && defined(ENABLE_PSI_CLASSICAL_ANN)
template void TDVP::eval(const Operator&, const PsiClassicalANN<2u>&, MonteCarlo_tt<PauliString>&);
#endif
#if defined(ENABLE_EXACT_SUMMATION) && defined(ENABLE_SPINS) && defined(ENABLE_PSI_CLASSICAL)
template void TDVP::eval(const Operator&, const PsiClassicalFP<1u>&, ExactSummation_t<Spins>&);
#endif
#if defined(ENABLE_EXACT_SUMMATION) && defined(ENABLE_SPINS) && defined(ENABLE_PSI_CLASSICAL)
template void TDVP::eval(const Operator&, const PsiClassicalFP<2u>&, ExactSummation_t<Spins>&);
#endif
#if defined(ENABLE_EXACT_SUMMATION) && defined(ENABLE_SPINS) && defined(ENABLE_PSI_CLASSICAL) && defined(ENABLE_PSI_CLASSICAL_ANN)
template void TDVP::eval(const Operator&, const PsiClassicalANN<1u>&, ExactSummation_t<Spins>&);
#endif
#if defined(ENABLE_EXACT_SUMMATION) && defined(ENABLE_SPINS) && defined(ENABLE_PSI_CLASSICAL) && defined(ENABLE_PSI_CLASSICAL_ANN)
template void TDVP::eval(const Operator&, const PsiClassicalANN<2u>&, ExactSummation_t<Spins>&);
#endif
#if defined(ENABLE_EXACT_SUMMATION) && defined(ENABLE_PAULIS) && defined(ENABLE_PSI_CLASSICAL)
template void TDVP::eval(const Operator&, const PsiClassicalFP<1u>&, ExactSummation_t<PauliString>&);
#endif
#if defined(ENABLE_EXACT_SUMMATION) && defined(ENABLE_PAULIS) && defined(ENABLE_PSI_CLASSICAL)
template void TDVP::eval(const Operator&, const PsiClassicalFP<2u>&, ExactSummation_t<PauliString>&);
#endif
#if defined(ENABLE_EXACT_SUMMATION) && defined(ENABLE_PAULIS) && defined(ENABLE_PSI_CLASSICAL) && defined(ENABLE_PSI_CLASSICAL_ANN)
template void TDVP::eval(const Operator&, const PsiClassicalANN<1u>&, ExactSummation_t<PauliString>&);
#endif
#if defined(ENABLE_EXACT_SUMMATION) && defined(ENABLE_PAULIS) && defined(ENABLE_PSI_CLASSICAL) && defined(ENABLE_PSI_CLASSICAL_ANN)
template void TDVP::eval(const Operator&, const PsiClassicalANN<2u>&, ExactSummation_t<PauliString>&);
#endif


} // namespace ann_on_gpu


#endif // LEAN_AND_MEAN
