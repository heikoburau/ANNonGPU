#include "hip/hip_runtime.h"
#include "network_functions/ExpectationValue.hpp"
#include "ensembles.hpp"
#include "quantum_states.hpp"
#include "Array.hpp"


namespace ann_on_gpu {

ExpectationValue::ExpectationValue(const bool gpu) : A_local(1, gpu), A_local_abs2(1, gpu) {
}

template<typename Psi_t, typename Ensemble>
complex<double> ExpectationValue::operator()(
    const Psi_t& psi, const Operator& operator_, Ensemble& ensemble
) {

    this->A_local.clear();
    auto A_local_ptr = this->A_local.data();
    auto psi_kernel = psi.kernel();
    auto op_kernel = operator_.kernel();

    ensemble.foreach(
        psi,
        [=] __device__ __host__ (
            const unsigned int spin_index,
            const typename Ensemble::Basis_t& basis_vector,
            const typename Psi_t::dtype log_psi,
            typename Psi_t::dtype* angles,
            typename Psi_t::dtype* activations,
            const typename Psi_t::real_dtype weight
        ) {
            #include "cuda_kernel_defines.h"

            SHARED typename Psi_t::dtype local_energy;
            op_kernel.local_energy(local_energy, psi_kernel, basis_vector, log_psi, angles, activations);

            SINGLE {
                generic_atomicAdd(A_local_ptr, weight * local_energy);
            }
        }
    );

    this->A_local.update_host();
    return this->A_local.front().to_std();
}


template<typename Psi_t, typename Ensemble>
pair<double, complex<double>> ExpectationValue::fluctuation(
    const Psi_t& psi, const Operator& operator_, Ensemble& ensemble
) {

    this->A_local.clear();
    this->A_local_abs2.clear();

    auto A_local_ptr = this->A_local.data();
    auto A_local_abs2_ptr = this->A_local_abs2.data();
    auto psi_kernel = psi.kernel();
    auto op_kernel = operator_.kernel();

    ensemble.foreach(
        psi,
        [=] __device__ __host__ (
            const unsigned int spin_index,
            const typename Ensemble::Basis_t& basis_vector,
            const typename Psi_t::dtype log_psi,
            typename Psi_t::dtype* angles,
            typename Psi_t::dtype* activations,
            const typename Psi_t::real_dtype weight
        ) {
            #include "cuda_kernel_defines.h"

            SHARED typename Psi_t::dtype local_energy;
            op_kernel.local_energy(local_energy, psi_kernel, basis_vector, log_psi, angles, activations);

            SINGLE {
                generic_atomicAdd(A_local_ptr, weight * local_energy);
                generic_atomicAdd(A_local_abs2_ptr, weight * abs2(local_energy));
            }
        }
    );

    this->A_local.update_host();
    this->A_local_abs2.update_host();

    return {
        sqrt(this->A_local_abs2.front() - abs2(this->A_local.front())),
        this->A_local.front().to_std()
    };
}


#ifdef ENABLE_MONTE_CARLO

template complex<double> ExpectationValue::operator()(const PsiDeep& psi, const Operator&, MonteCarloLoop&);
template pair<double, complex<double>> ExpectationValue::fluctuation(const PsiDeep&, const Operator&, MonteCarloLoop&);

#endif // ENABLE_MONTE_CARLO

#ifdef ENABLE_MONTE_CARLO_PAULIS

template complex<double> ExpectationValue::operator()(const PsiDeep& psi, const Operator&, MonteCarloLoopPaulis&);
template pair<double, complex<double>> ExpectationValue::fluctuation(const PsiDeep&, const Operator&, MonteCarloLoopPaulis&);

#endif // ENABLE_MONTE_CARLO_PAULIS

#ifdef ENABLE_EXACT_SUMMATION

template complex<double> ExpectationValue::operator()(const PsiDeep& psi, const Operator&, ExactSummation&);
template pair<double, complex<double>> ExpectationValue::fluctuation(const PsiDeep&, const Operator&, ExactSummation&);

#endif // ENABLE_EXACT_SUMMATION

} // namespace ann_on_gpu
