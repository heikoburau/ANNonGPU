
#include <hip/hip_runtime.h>
// #include "network_functions/PsiOkVector.hpp"
// #include "quantum_states.hpp"
// #include "ensembles.hpp"
// #include "types.h"

// namespace ann_on_gpu {


// template<typename Psi_t>
// void psi_O_k_vector(complex<double>* result, const Psi_t& psi, const Spins& spins) {
//     complex_t* result_ptr;
//     auto O_k_length = psi.num_params;
//     auto psi_kernel = psi.kernel();

//     // printf("O_k_length: %d\n", O_k_length);
//     MALLOC(result_ptr, sizeof(complex_t) * O_k_length, psi.gpu);
//     // MEMSET(result_ptr, 0, sizeof(complex_t) * O_k_length, psi.gpu);

//     const auto functor = [=] __host__ __device__ () {
//         #include "cuda_kernel_defines.h"

//         SHARED typename Psi_t::dtype activations[Psi_t::max_width];

//         psi_kernel.foreach_O_k(
//             spins,
//             activations,
//             [&](const unsigned int k, const typename Psi_t::dtype& O_k_element) {
//                 // printf("%d, %f, %f\n", k, O_k_element.real(), O_k_element.imag());
//                 result_ptr[k] = precision_cast<complex_t>(O_k_element);
//             }
//         );
//     };

//     if(psi.gpu) {
//         cuda_kernel<<<1, psi.get_width()>>>(functor);
//     }
//     else {
//         functor();
//     }

//     MEMCPY_TO_HOST(result, result_ptr, sizeof(complex_t) * O_k_length, psi.gpu);
//     FREE(result_ptr, psi.gpu);
// }


// template<typename Psi_t, typename SpinEnsemble>
// void psi_O_k_vector(complex<double>* result, complex<double>* result_std, const Psi_t& psi, SpinEnsemble& spin_ensemble) {
//     const auto O_k_length = psi.num_params;
//     const auto psi_kernel = psi.kernel();

//     complex_t* result_device;
//     complex_t* result2_device;

//     MALLOC(result_device, sizeof(complex_t) * O_k_length, psi.gpu);
//     MALLOC(result2_device, sizeof(complex_t) * O_k_length, psi.gpu);
//     MEMSET(result_device, 0, sizeof(complex_t) * O_k_length, psi.gpu);
//     MEMSET(result2_device, 0, sizeof(complex_t) * O_k_length, psi.gpu);

//     spin_ensemble.foreach(
//         psi,
//         [=] __device__ __host__ (
//             const unsigned int spin_index,
//             const Spins spins,
//             const complex_t log_psi,
//             typename Psi_t::dtype* angles,
//             typename Psi_t::dtype* activations,
//             const double weight
//         ) {
//             psi_kernel.foreach_O_k(
//                 spins,
//                 activations,
//                 [&](const unsigned int k, const complex_t& O_k_element) {
//                     generic_atomicAdd(&result_device[k], weight * O_k_element);
//                     const auto O_k_element2 = complex_t(
//                         O_k_element.real() * O_k_element.real(), O_k_element.imag() * O_k_element.imag()
//                     );
//                     generic_atomicAdd(&result2_device[k], weight * O_k_element2);
//                 }
//             );
//         }
//     );

//     MEMCPY_TO_HOST(result, result_device, sizeof(complex_t) * O_k_length, psi.gpu);
//     MEMCPY_TO_HOST(result_std, result2_device, sizeof(complex_t) * O_k_length, psi.gpu);
//     FREE(result_device, psi.gpu);
//     FREE(result2_device, psi.gpu);

//     for(auto k = 0u; k < O_k_length; k++) {
//         result[k] /= spin_ensemble.get_num_steps();
//         result_std[k] /= spin_ensemble.get_num_steps();

//         result_std[k] = result_std[k] - complex<double>(
//             result[k].real() * result[k].real(), result[k].imag() * result[k].imag()
//         );
//     }
// }


// template<typename Psi_t, typename SpinEnsemble>
// pair<Array<complex_t>, Array<double>> psi_O_k_vector(const Psi_t& psi, SpinEnsemble& spin_ensemble) {
//     const auto O_k_length = psi.num_params;
//     const auto psi_kernel = psi.kernel();

//     Array<complex_t> result(O_k_length, psi.gpu);
//     Array<double> result_std(O_k_length, psi.gpu);

//     result.clear();
//     result_std.clear();

//     auto result_ptr = result.data();
//     auto result_std_ptr = result_std.data();

//     spin_ensemble.foreach(
//         psi,
//         [=] __device__ __host__ (
//             const unsigned int spin_index,
//             const Spins spins,
//             const complex_t log_psi,
//             typename Psi_t::dtype* angles,
//             typename Psi_t::dtype* activations,
//             const double weight
//         ) {
//             psi_kernel.foreach_O_k(
//                 spins,
//                 activations,
//                 [&](const unsigned int k, const complex_t& O_k_element) {
//                     generic_atomicAdd(&result_ptr[k], weight * O_k_element);
//                     generic_atomicAdd(&result_std_ptr[k], weight * (O_k_element * conj(O_k_element)).real());
//                 }
//             );
//         }
//     );

//     result.update_host();
//     result_std.update_host();

//     for(auto k = 0u; k < O_k_length; k++) {
//         result[k] /= spin_ensemble.get_num_steps();
//         result_std[k] /= spin_ensemble.get_num_steps();

//         result_std[k] = sqrt((result_std[k] - result[k] * conj(result[k])).real());
//     }

//     return {result, result_std};
// }

// #ifdef ENABLE_PSI_DEEP
// template void psi_O_k_vector(complex<double>* result, const PsiDeep& psi, const Spins& spins);
// #endif // ENABLE_PSI_DEEP

// #ifdef ENABLE_PSI_PAIR
// // template void psi_O_k_vector(complex<double>* result, const PsiPair& psi, const Spins& spins);
// #endif // ENABLE_PSI_PAIR


// } // namespace ann_on_gpu
