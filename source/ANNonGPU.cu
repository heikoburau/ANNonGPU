#include "types.h"
#include <hip/hip_runtime_api.h>


namespace ann_on_gpu {

void setDevice(int device) {
    hipSetDevice(device);
}

void start_profiling() {
    hipProfilerStart();
}

void stop_profiling() {
    hipProfilerStop();
}

} // namespace ann_on_gpu
